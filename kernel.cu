
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512


__global__ void naiveReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    // NAIVE REDUCTION IMPLEMENTATION

	__shared__ float partialSum[2 * BLOCK_SIZE];

    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;

    if (start + t < size) {
        partialSum[t] = in[start + t];
    } else {
        partialSum[t] = 0.0f;  
    }

    if (start + blockDim.x + t < size) {
        partialSum[blockDim.x + t] = in[start + blockDim.x + t];
    } else {
        partialSum[blockDim.x + t] = 0.0f;  
    }

    __syncthreads();  

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        if (t % stride == 0) {
            partialSum[2 * t] += partialSum[2 * t + stride];
        }
    }

    __syncthreads();  

    if (t == 0) {
        out[blockIdx.x] = partialSum[0];  
    }
}

__global__ void optimizedReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    // OPTIMIZED REDUCTION IMPLEMENTATION
    __shared__ float partialSum[2 * BLOCK_SIZE];

    unsigned int t = threadIdx.x; 
    unsigned int start = 2 * blockIdx.x * blockDim.x;  

    if (start + t < size) {
        partialSum[t] = in[start + t];
    } else {
        partialSum[t] = 0.0f;  
    }

    if (start + blockDim.x + t < size) {
        partialSum[blockDim.x + t] = in[start + blockDim.x + t];
    } else {
        partialSum[blockDim.x + t] = 0.0f;  
    }

    __syncthreads();

    for (unsigned int stride = blockDim.x; stride > 0; stride >>= 1) {
        if (t < stride) {
            partialSum[t] += partialSum[t + stride];
        }
        __syncthreads();
    }

    if (t == 0) {
        out[blockIdx.x] = partialSum[0];
    }
}



