
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512


__global__ void naiveReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    // NAIVE REDUCTION IMPLEMENTATION

	__shared__ float sdata[BLOCK_SIZE];

	unsigned int tid = threadIdx.x;
   	unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

	if (i < size) {
        sdata[tid] = in[i] + in[i + blockDim.x];
   	 } else {
        sdata[tid] = 0;
    	}
   	__syncthreads();
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            sdata[tid] += sdata[tid + s];
        }
	__syncthreads();
    }
	if (tid == 0) {
        out[blockIdx.x] = sdata[0];
    }
}

__global__ void optimizedReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    // OPTIMIZED REDUCTION IMPLEMENTATION
     __shared__ float sdata[BLOCK_SIZE]; 

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    if (i < size) {
        sdata[tid] = in[i] + in[i + blockDim.x];
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();  

    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1){
        if (tid < s) {
 
           sdata[tid] += sdata[tid + s];
        }
 	__syncthreads();
     }
 if (tid < 32) {
    volatile float* vsmem = sdata;
    vsmem[tid] += vsmem[tid + 32];
    vsmem[tid] += vsmem[tid + 16];
    vsmem[tid] += vsmem[tid + 8];
    vsmem[tid] += vsmem[tid + 4];
    vsmem[tid] += vsmem[tid + 2];
    vsmem[tid] += vsmem[tid + 1];
}   

     if (tid == 0) {
        out[blockIdx.x] = sdata[0];
     }
}



